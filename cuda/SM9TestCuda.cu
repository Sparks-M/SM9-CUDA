#include "SM9Cuda.cuh"

float SM9TEST_INIT_CUDA(int N, int numBlock, int blockSize, struct SM9_Para *gpu_para)
{
    hipEvent_t start, end;
    float time, res = 0;

    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    SM9_Init_CUDA<<<numBlock, blockSize>>>(N, gpu_para);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("\nInit_CUDA计时结果: %.6f ms\n", time);
    res += time;

    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    SM9_GenerateSignKey_CUDA<<<numBlock, blockSize>>>(N, gpu_para);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("\nGenerateSignKey_CUDA计时结果: %.6f ms\n", time);
    res += time;

    return res;
}

float SM9TEST_SIGN_CUDA(int N, int numBlock, int blockSize, struct SM9_Para *gpu_para)
{
    hipEvent_t start, end;
    float time, res = 0;

    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    SM9_Sign_Init_CUDA<<<numBlock, blockSize>>>(N, gpu_para);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("\nSM9_Sign_Init_CUDA耗时: %.6f ms\n", time);
    res += time;

    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    SM9_Sign_Step1_CUDA<<<numBlock, blockSize>>>(N, gpu_para); // Step1:g = e(P1, Ppub-s)
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("\nSM9_Sign_Step1_CUDA耗时: %.6f ms\n", time);
    res += time;

    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    SM9_Sign_Step2_CUDA<<<numBlock, blockSize>>>(N, gpu_para); // Step2:calculate w=g(r)
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("\nSM9_Sign_Step2_CUDA耗时: %.6f ms\n", time);
    res += time;

    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    SM9_Sign_Step3_CUDA<<<numBlock, blockSize>>>(N, gpu_para); // Step3:calculate h=H2(M||w,N)
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("\nSM9_Sign_Step3_CUDA耗时: %.6f ms\n", time);
    res += time;

    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    SM9_Sign_Step4_CUDA<<<numBlock, blockSize>>>(N, gpu_para); // Step4:l=(r-h)mod N
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("\nSM9_Sign_Step4_CUDA耗时: %.6f ms\n", time);
    res += time;

    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    SM9_Sign_Step5_CUDA<<<numBlock, blockSize>>>(N, gpu_para); // Step5:S=[l]dSA=(xS,yS)
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("\nSM9_Sign_Step5_CUDA耗时: %.6f ms\n", time);
    res += time;

    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    SM9_Sign_Finish_CUDA<<<numBlock, blockSize>>>(N, gpu_para);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("\nSM9_Sign_Finish_CUDA耗时: %.6f ms\n", time);
    res += time;

    return res;
}

//struct SM9_Para cpu_para[1024];

float SM9TEST_VERIFY_CUDA(int N, int numBlock, int blockSize, struct SM9_Para *gpu_para)
{
    hipEvent_t start, end;
    float time, res = 0;

    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    SM9_Verify_Init_CUDA<<<numBlock, blockSize>>>(N, gpu_para);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("\nSM9_Verify_Init_CUDA耗时: %.6f ms\n", time);
    res += time;

    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    SM9_Verify_Step1_CUDA<<<numBlock, blockSize>>>(N, gpu_para); // Step 1:test if h in the rangge [1,N-1]
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("\nSM9_Verify_Step1_CUDA耗时: %.6f ms\n", time);
    res += time;

    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    SM9_Verify_Step2_CUDA<<<numBlock, blockSize>>>(N, gpu_para); // Step 2:test if S is on G1
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("\nSM9_Verify_Step2_CUDA耗时: %.6f ms\n", time);
    res += time;

    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    SM9_Verify_Step3_CUDA<<<numBlock, blockSize>>>(N, gpu_para); // Step3:g = e(P1, Ppub-s)
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("\nSM9_Verify_Step3_CUDA耗时: %.6f ms\n", time);
    res += time;

    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    SM9_Verify_Step4_CUDA<<<numBlock, blockSize>>>(N, gpu_para); // Step4:calculate t=g^h
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("\nSM9_Verify_Step4_CUDA耗时: %.6f ms\n", time);
    res += time;

    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    SM9_Verify_Step5_CUDA<<<numBlock, blockSize>>>(N, gpu_para); // Step5:calculate h1=H1(IDA||hid,N)
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("\nSM9_Verify_Step5_CUDA耗时: %.6f ms\n", time);
    res += time;

    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    SM9_Verify_Step6_CUDA<<<numBlock, blockSize>>>(N, gpu_para); // Step6:P=[h1]P2+Ppubs
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("\nSM9_Verify_Step6_CUDA耗时: %.6f ms\n", time);
    res += time;

//     for (int i = 0; i < N; i++)
//         ParaMemcpy2Host(cpu_para + i, gpu_para + i);
//     hipDeviceReset();
//     hipDeviceSetLimit(hipLimitMallocHeapSize, N * CUDA_HEAP_SIZE);
//     hipMalloc((void **)&gpu_para, N * sizeof(struct SM9_Para));
//     for (int i = 0; i < N; i++)
//         ParaMemcpy2Device(gpu_para + i, cpu_para + i);

    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    SM9_Verify_Step7_CUDA<<<numBlock, blockSize>>>(N, gpu_para); // Step7:u=e(S1,P)
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("\nSM9_Verify_Step7_CUDA耗时: %.6f ms\n", time);
    res += time;

    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    SM9_Verify_Step8_CUDA<<<numBlock, blockSize>>>(N, gpu_para); // Step8:w=u*t
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("\nSM9_Verify_Step8_CUDA耗时: %.6f ms\n", time);
    res += time;

    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    SM9_Verify_Step9_CUDA<<<numBlock, blockSize>>>(N, gpu_para); // Step9:h2=H2(M||w,N)
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("\nSM9_Verify_Step9_CUDA耗时: %.6f ms\n", time);
    res += time;

    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start, 0);
    SM9_Verify_Finish_CUDA<<<numBlock, blockSize>>>(N, gpu_para);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("\nSM9_Verify_Finish_CUDA耗时: %.6f ms\n", time);
    res += time;

    return res;
}

int SM9TEST_CUDA(int N, int blockSize, struct SM9_Para *para)
{
//    struct SM9_Para *cpu_para; // malloc(Heap//    cpu_para = (struct SM9_Para *)malloc(N * sizeof(SM9_Para));) instead of cpu_para[N](Stack)

//    for (int i = 0; i < N; i++)
//        memcpy(cpu_para + i, para, sizeof(SM9_Para));

    // 设置显存堆大小为CUDA_HEAP_SIZE
    hipDeviceSetLimit(hipLimitMallocHeapSize, N * CUDA_HEAP_SIZE);

    // =====计时开始=====
    hipEvent_t copy_start, copy_end;
    hipEventCreate(&copy_start);
    hipEventCreate(&copy_end);
    hipEventRecord(copy_start, 0);
    // =====申请显存并拷贝参数开始=====
    struct SM9_Para *gpu_para;
    hipMalloc((void **)&gpu_para, N * sizeof(struct SM9_Para));
    for (int i = 0; i < N; i++)
         hipMemcpy(gpu_para + i, para, sizeof(struct SM9_Para), hipMemcpyHostToDevice);
//        ParaMemcpy2Device(gpu_para + i, cpu_para + i);

    // printf("%s\n", para->SM9_q);
    // =====申请显存并拷贝参数结束=====
    // =====计时结束=====
    hipEventRecord(copy_end, 0);
    hipEventSynchronize(copy_end);
    float copy_time;
    hipEventElapsedTime(&copy_time, copy_start, copy_end);
    printf("\ngpu显存拷贝耗时: %.6f ms\n", copy_time);

    int numBlock = (N + blockSize - 1) / blockSize;

    { // SM9TEST_INIT_CUDA
        SM9TEST_INIT_CUDA(N, numBlock, blockSize, gpu_para);
    }

//    for (int i = 0; i < N; i++)
//        ParaMemcpy2Host(cpu_para + i, gpu_para + i);
//    hipDeviceReset();
//    hipDeviceSetLimit(hipLimitMallocHeapSize, N * CUDA_HEAP_SIZE);
//    hipMalloc((void **)&gpu_para, N * sizeof(struct SM9_Para));
//    for (int i = 0; i < N; i++)
//        ParaMemcpy2Device(gpu_para + i, cpu_para + i);
    
//    Print_Para(cpu_para);

    { // SM9TEST_SIGN_CUDA
        float time = SM9TEST_SIGN_CUDA(N, numBlock, blockSize, gpu_para);

        printf("\nsign_cuda计时结果: %.6f ms\n", time);
    }

//     for (int i = 0; i < N; i++)
//         ParaMemcpy2Host(cpu_para + i, gpu_para + i);
//
////     Print_Para(cpu_para + i);
//
//     hipDeviceReset();
//     hipDeviceSetLimit(hipLimitMallocHeapSize, N * CUDA_HEAP_SIZE);
//     hipMalloc((void **)&gpu_para, N * sizeof(struct SM9_Para));
//     for (int i = 0; i < N; i++)
//         ParaMemcpy2Device(gpu_para + i, cpu_para + i);

    { // SM9TEST_VERIFY_CUDA
        float time = SM9TEST_VERIFY_CUDA(N, numBlock, blockSize, gpu_para);

        printf("\nverify_cuda计时结果: %.6f ms\n", time);
    }
    // ParaMemcpy(&cpu_para, gpu_para, hipMemcpyDeviceToHost);
    // hipDeviceReset();

    return 0;
}
